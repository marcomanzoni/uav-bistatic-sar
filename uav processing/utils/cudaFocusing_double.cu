#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <iostream>

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        mexErrMsgTxt("CUDA Error,exit.");
    }
}

__device__ double gaussActivFunc(double x, double sigma)
{
    double out = exp(-0.5 * (x / sigma) * (x / sigma));
    return out;
}

__device__ double2 linear_interp_comp(double const *x, double2 const *y, double const xq, int const y_zero_idx, int const N)
{
    // RC is the whole matrix, has to take only 1 column
    int const y_end_idx = y_zero_idx + N;

    // Manage extrapolation
    if (xq <= x[0])
        return y[0];
    if (xq >= x[N - 1])
        return y[y_end_idx - 1];
    double2 yout;
    // search the left value
    int j = 0;
    while (x[j + 1] < xq)
        j++;
    double xL = x[j], xR = x[j + 1];
    double2 yL = y[y_zero_idx + j], yR = y[y_zero_idx + j + 1];
    double grad_real = (yR.x - yL.x) / (xR - xL);
    double grad_imag = (yR.y - yL.y) / (xR - xL);
    yout.x = yL.x + grad_real * (xq - xL);
    yout.y = yL.y + grad_imag * (xq - xL);
    return yout;
}

__device__ double2 exp_comp(double const ampl, double const phase)
{
    double2 out;
    out.x = ampl * cos(phase);
    out.y = ampl * sin(phase);
    return out;
}

__device__ double2 mult_comp(double2 const a, double2 const b)
{
    double2 out;
    out.x = a.x * b.x - a.y * b.y;
    out.y = a.x * b.y + a.y * b.x;
    return out;
}
__global__ void focusTDBPKernel(double const *X, double const *Y, double const z0, double const *TX_pos_x,
                                double const *TX_pos_y, double const *TX_pos_z, double const *RX_pos_x, double const *RX_pos_y, double const *RX_pos_z,
                                double const lambda, double const Dk, double2 const *RC, double const *t, double const f0, double const k_rx_0,
                                double2 *Sn, double *Wn, int const N_pixel, int const N_RC, int const tau, int const squint, double const speed_weight)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N_pixel)
        return;
    int out_i = i + N_pixel*squint;
    
    double X_i = X[i];
    double RX_pos_x_tau = RX_pos_x[tau];
    if (X_i < RX_pos_x_tau)
    {
        // Backlobe of antenna, pixel is 0
        Wn[out_i] += 0.0;
        Sn[out_i].x += 0.0;
        Sn[out_i].y += 0.0;
        return;
    };
    double const C = 2.99792458e8;
    double const pi = 3.1415926535897932385;
    double Y_i = Y[i];
    double TX_pos_x_tau = TX_pos_x[tau];
    double TX_pos_y_tau = TX_pos_y[tau];
    double TX_pos_z_tau = TX_pos_z[tau];
    double RX_pos_y_tau = RX_pos_y[tau];
    double RX_pos_z_tau = RX_pos_z[tau];

    // Range distances from the tx antenna [m]
    double R_tx = sqrt((TX_pos_x_tau - X_i) * (TX_pos_x_tau - X_i) + (TX_pos_y_tau - Y_i) * (TX_pos_y_tau - Y_i) + (TX_pos_z_tau - z0) * (TX_pos_z_tau - z0));
    // Range distances from the rx antenna [m]
    double R_rx = sqrt((RX_pos_x_tau - X_i) * (RX_pos_x_tau - X_i) + (RX_pos_y_tau - Y_i) * (RX_pos_y_tau - Y_i) + (RX_pos_z_tau - z0) * (RX_pos_z_tau - z0));
    // Total Tx-target-Rx distance [m]
    double distance = R_tx + R_rx;
    double delay = distance / C;

    // Compute target wave number
    double R = sqrt((RX_pos_x_tau - X_i) * (RX_pos_x_tau - X_i) + (RX_pos_y_tau - Y_i) * (RX_pos_y_tau - Y_i));
    double psi = asin((Y_i - RX_pos_y_tau) / R);
    double k_rx = sin(psi) * 2 * pi / lambda;

    // Weight function
    double sigma = Dk / 2;
    
    double Wn_i = speed_weight * gaussActivFunc(k_rx - k_rx_0, sigma);

    // Backprojection of data from a single Radar position
    int const RC_zero_idx = tau * N_RC;
    double2 RC_1 = linear_interp_comp(t, RC, delay, RC_zero_idx, N_RC);

    double2 RC_2 = mult_comp(RC_1, exp_comp(1, 2 * pi * f0 * delay));
    Sn[out_i].x += Wn_i * RC_2.x;
    Sn[out_i].y += Wn_i * RC_2.y;
    Wn[out_i] += Wn_i;
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
    //========================================= Input variables
    mxGPUArray const *X, *Y, *RC, *t, *TX_pos_x, *TX_pos_y, *TX_pos_z, *RX_pos_x, *RX_pos_y, *RX_pos_z;

    double const *d_X, *d_Y, *d_t, *d_TX_pos_x, *d_TX_pos_y, *d_TX_pos_z, *d_RX_pos_x, *d_RX_pos_y, *d_RX_pos_z,*k_rx_0_vec,*RX_speed;
    double2 const *d_RC;
    //========================================= Constants
    double const *_z0, *_lambda, *_Dk, *_f0,*_median_speed;

    //========================================= Output variables
    mxGPUArray *Sn, *Wn;
    double2 *d_Sn;
    double *d_Wn;
    
    int N_pixel, N_RC, N_tau, N_squints;
    char const *const errId = "parallel:gpu:mexGPUExample:InvalidInput";

    //========================================= Input validation
    if (nrhs != 17 || nlhs != 2)
        mexErrMsgIdAndTxt(errId, "Wrong number of input/output arguments.");
    if (!(mxIsGPUArray(prhs[0]) && mxIsGPUArray(prhs[1])))
        mexErrMsgIdAndTxt(errId, "Input must be GPUArray");

    int const threadsPerBlock = 256;
    int blocksPerGrid;

    mxInitGPU();
    //========================================= Initialize variables
    X = mxGPUCreateFromMxArray(prhs[0]);
    Y = mxGPUCreateFromMxArray(prhs[1]);
    _z0 = (double const *)mxGetData(prhs[2]);
    TX_pos_x = mxGPUCreateFromMxArray(prhs[3]);
    TX_pos_y = mxGPUCreateFromMxArray(prhs[4]);
    TX_pos_z = mxGPUCreateFromMxArray(prhs[5]);
    RX_pos_x = mxGPUCreateFromMxArray(prhs[6]);
    RX_pos_y = mxGPUCreateFromMxArray(prhs[7]);
    RX_pos_z = mxGPUCreateFromMxArray(prhs[8]);
    _lambda = (double const *)mxGetData(prhs[9]);
    _Dk = (double const *)mxGetData(prhs[10]);
    RC = mxGPUCreateFromMxArray(prhs[11]);
    t = mxGPUCreateFromMxArray(prhs[12]);
    _f0 = (double const *)mxGetData(prhs[13]);
    k_rx_0_vec = (double const *) mxGetData(prhs[14]);
    RX_speed = (double const *) mxGetData(prhs[15]);
    _median_speed = (double const *) mxGetData(prhs[16]);

    double const z0 = _z0[0];
    double const lambda = _lambda[0];
    double const Dk = _Dk[0];
    double const f0 = _f0[0];
    double const median_speed = _median_speed[0];

    if (mxGPUGetClassID(X) != mxDOUBLE_CLASS)
    {
        mexErrMsgIdAndTxt(errId, "Input must be double");
    }

    //========================================= Initialize pointers
    d_X = (double const *)mxGPUGetDataReadOnly(X);
    d_Y = (double const *)mxGPUGetDataReadOnly(Y);
    d_TX_pos_x = (double const *)mxGPUGetDataReadOnly(TX_pos_x);
    d_TX_pos_y = (double const *)mxGPUGetDataReadOnly(TX_pos_y);
    d_TX_pos_z = (double const *)mxGPUGetDataReadOnly(TX_pos_z);
    d_RX_pos_x = (double const *)mxGPUGetDataReadOnly(RX_pos_x);
    d_RX_pos_y = (double const *)mxGPUGetDataReadOnly(RX_pos_y);
    d_RX_pos_z = (double const *)mxGPUGetDataReadOnly(RX_pos_z);
    d_RC = (double2 const *)mxGPUGetDataReadOnly(RC);
    d_t = (double const *)mxGPUGetDataReadOnly(t);

    //========================================= Create ouput array
    N_pixel = (int)mxGPUGetNumberOfElements(X);
    N_RC = (int)mxGPUGetDimensions(RC)[0];
    N_tau = (int)mxGPUGetDimensions(RC)[1];
    N_squints = (int)mxGetDimensions(prhs[14])[0];
    
    mwSize out_N_dim = 3, out_dims[3] = {mxGPUGetDimensions(X)[0],mxGPUGetDimensions(X)[1],N_squints}; 
    
    Sn = mxGPUCreateGPUArray(out_N_dim,
                             out_dims,
                             mxGPUGetClassID(RC),
                             mxGPUGetComplexity(RC),
                             MX_GPU_INITIALIZE_VALUES);
    d_Sn = (double2 *)mxGPUGetData(Sn);
    Wn = mxGPUCreateGPUArray(out_N_dim,
                             out_dims,
                             mxGPUGetClassID(X),
                             mxGPUGetComplexity(X),
                             MX_GPU_INITIALIZE_VALUES);
    d_Wn = (double *)mxGPUGetData(Wn);

    //========================================= Elaboration

    
    std::cout << "\nN_pix " << N_pixel << ", N_RC " << N_RC << "\n";
    std::cout << "lambda " << lambda << " freq " << f0 << "\n";
    std::cout << "RC dim 1 " << N_RC << "\n";
    std::cout << "RC dim 2 " << N_tau << "\n";
    std::cout << "N squints " << N_squints << "\n";
    
    blocksPerGrid = (N_pixel + threadsPerBlock - 1) / threadsPerBlock;
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    double speed_weight;
    for(int squint = 0; squint < N_squints;squint++){
        for (int tau = 0; tau < N_tau; tau++)
        {
            speed_weight = RX_speed[tau] / median_speed;
            focusTDBPKernel<<<blocksPerGrid, threadsPerBlock>>>(
                d_X, d_Y, z0, d_TX_pos_x, d_TX_pos_y, d_TX_pos_z, d_RX_pos_x, d_RX_pos_y, d_RX_pos_z,
                lambda, Dk, d_RC, d_t, f0, k_rx_0_vec[squint], d_Sn, d_Wn, N_pixel, N_RC, tau, squint, speed_weight);
        }
        std::cout << "Squint k_rx " << k_rx_0_vec[squint] << "\n";
        std::cout << "Squint n " << squint +1 << " of " << N_squints << "\n";
    }
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // OUTPUT

    plhs[0] = mxGPUCreateMxArrayOnGPU(Sn);
    plhs[1] = mxGPUCreateMxArrayOnGPU(Wn);

    // Destroy
    mxGPUDestroyGPUArray(Sn);
    mxGPUDestroyGPUArray(Wn);
    mxGPUDestroyGPUArray(X);
    mxGPUDestroyGPUArray(Y);
    mxGPUDestroyGPUArray(TX_pos_x);
    mxGPUDestroyGPUArray(TX_pos_y);
    mxGPUDestroyGPUArray(TX_pos_z);
    mxGPUDestroyGPUArray(RX_pos_x);
    mxGPUDestroyGPUArray(RX_pos_y);
    mxGPUDestroyGPUArray(RX_pos_z);
    mxGPUDestroyGPUArray(RC);
    mxGPUDestroyGPUArray(t);
    //mxGPUDestroyGPUArray(k_rx_0_vec);
}